#include <stdio.h>
#include <hip/hip_runtime.h>
#define TILE_SIZE 16
__global__ void matrix_multiplication(float* a, float* b, float* c, int M, int N){
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    __shared__ float tile_a[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_b[TILE_SIZE][TILE_SIZE];
    float sum = 0.0;
    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        if (row < M && t * TILE_SIZE + tx < N) {
            tile_a[ty][tx] = a[row * N + t * TILE_SIZE + tx];}
            else {
            tile_a[ty][tx] = 0.0;}
        if (col < N && t * TILE_SIZE + ty < N) {
            tile_b[ty][tx] = b[(t * TILE_SIZE + ty) * N + col];} 
        else {
            tile_b[ty][tx] = 0.0;}
        __syncthreads();
        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += tile_a[ty][k] * tile_b[k][tx];}
        __syncthreads();}
    if (row < M && col < N) {
        c[row * N + col] = sum;}
}
int main(){
    int M = 1000;
    int N = 500;
    float* a, * b, * c;
    float* d_a, * d_b, * d_c;
    int size_a = M * N * sizeof(float);
    int size_b = N * N * sizeof(float);
    int size_c = M * N * sizeof(float);
    a = (float*)malloc(size_a);
    b = (float*)malloc(size_b);
    c = (float*)malloc(size_c);
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            a[i * N + j] = i + j;}
    }
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            b[i * N + j] = i - j;}
    }
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);
    hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice);
    dim3 dimGrid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE, 1);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    matrix_multiplication<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, M, N);
    hipEventRecord(stop);
    hipDeviceSynchronize();
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", c[i * N + j]);
        }
        printf("\n");
    }
    printf("Execution time: %.2f ms\n", milliseconds);
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}